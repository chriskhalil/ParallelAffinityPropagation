#include <stdio.h>
#include "AffinityPropagation.h"

using namespace Clustering;

void AffinityPropagation::buildGraph(FILE* input) {
    hipDeviceSynchronize();

    _graph = new Graph;
    fscanf_s(input, "%d", &_graph->n);
    _graph->outEdges = new Edges[_graph->n];
    _graph->inEdges = new Edges[_graph->n];
    vector<Edge>& edges = _graph->edges;

    
}