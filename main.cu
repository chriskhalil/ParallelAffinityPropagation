#include "hip/hip_runtime.h"
#include <stdio.h>
#include "AffinityPropagation.h"

void AffinityPropagation::update(double& variable, double newValue) {
    variable = _damping * variable + (1.0 - _damping) * newValue;
}


void AffinityPropagation::updateAvailabilities(){
    // for (int k{ 0 }; k < _graph->n; ++k) {
    //     Edges& edges{ _graph->inEdges[k] };
    //     size_t m{ edges.size() };
    //     // calculate sum of positive responsibilities
    //     auto sum{ 0.0 };
    //     for (int i{ 0 }; i < m - 1; ++i) {
    //         sum += max(0.0, edges[i]->responsability);
    //     }
    //     // calculate availabilities
    //     auto rkk{ edges[m - 1]->responsability };
    //     for (int i{ 0 }; i < m - 1; ++i) {
    //         update(edges[i]->availability, min(0.0, rkk + sum - max(0.0, edges[i]->responsability)));
    //     }
    //     // calculate self-availability
    //     update(edges[m - 1]->availability, sum);
    // }
    

        Edges* inEdges { _graph->inEdges }
        
    updateAvailabilities_GPU(inEdges);
}


void AffinityPropagation::updateResponsibilities() {
    // for (int i{ 0 }; i < _graph->n; ++i) {
    //     Edges& edges = _graph->outEdges[i];
    //     size_t m{ edges.size() };
    //     double max1{ -DBL_MAX }, max2{ -DBL_MAX };
    //     double argmax1{ -1 };
    //     for (int k{ 0 }; k < m; ++k) {
    //         double value{ edges[k]->similarity + edges[k]->availability };
    //         if (value > max1) { swap(max1, value); argmax1 = k; }
    //         if (value > max2) { max2 = value; }
    //     }
    //     // update responsibilities
    //     for (int k{ 0 }; k < m; ++k) {
    //         if (k != argmax1) {
    //             update(edges[k]->responsability, edges[k]->similarity - max1);
    //         }
    //         else {
    //             update(edges[k]->responsability, edges[k]->similarity - max2);
    //         }
    //     }
    // }

    hipDeviceSynchronize();
    // Edges& edges = _graph->outEdges[0];
    
}


// to do
void AffinityPropagation::buildGraph(FILE* input) {
    hipDeviceSynchronize();

    _graph = new Graph;
    fscanf_s(input, "%d", &_graph->n);
    _graph->outEdges = new Edges[_graph->n];
    _graph->inEdges = new Edges[_graph->n];
    vector<Edge>& edges = _graph->edges;

}